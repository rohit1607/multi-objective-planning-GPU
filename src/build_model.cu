#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/unique.h>
#include <thrust/count.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include "cnpy.h"
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <chrono>
using namespace std::chrono;
#include <iostream>
#include <sstream>
#include <fstream>


long long int ncells;


/*
------  Declarations of utility functions from utils.h -------
*/
cnpy::NpyArray read_velocity_field_data( std::string file_path_name, int* n_elements);
void define_xs_or_ys(float* xs, float dx, float x0, int gsize);
void save_master_Coos_to_file(std::string op_FnamePfx,std::string op_FnamePfx_2, int num_actions, 
thrust::host_vector<long long int> &H_master_cooS1, 
    thrust::host_vector<long long int> &H_master_cooS2, 
    thrust::host_vector<float> &H_master_cooVal,
    thrust::host_vector<float> &H_master_R,
    thrust::host_vector<long long int>* H_Aarr_of_cooS1,
    thrust::host_vector<long long int>* H_Aarr_of_cooS2,
    thrust::host_vector<float>* H_Aarr_of_cooProb,
    thrust::host_vector<float>* H_Aarr_of_Rs,
    thrust::host_vector<float> &prob_params,
    long long int* DP_relv_params,
    unsigned long int num_DP_params);


// template<typename dType> template not working for thrust vectors
void print_device_vector(thrust::device_vector<long long int> &array, int start_id, int end_id, std::string array_name, std::string end, int method);
void make_dir(std::string dir_name);
void populate_ac_angles(float* ac_angles, int num_ac_angles);
void populate_ac_speeds(float* ac_speeds, int num_ac_speeds, float Fmax);
void populate_actions(float** H_actions, int num_ac_speeds, int num_ac_angles, float Fmax);




/*
                    ----- Device functions -----
*/

__device__ int32_t get_thread_idx(){
    // assigns idx to thread with which it accesses the flattened 3d vxrzns matrix
    // for a given T and a given action. 
    // runs for both 2d and 3d grid
    // TODO: may have to change this considering cache locality
    // here i, j, k refer to a general matrix M[i][j][k]
    int32_t i = threadIdx.x;
    int32_t j = blockIdx.y;
    int32_t k = blockIdx.x;
    int32_t idx = k + (j*gridDim.x)  + (i*gridDim.x*gridDim.y)+ blockIdx.z*blockDim.x*gridDim.x*gridDim.y;
    return idx;
}

__device__ long long int state1D_from_spid(int32_t T, int32_t sp_id, long long int ncells){   
    // j ~ blockIdx.x
    // i ~ blockIdx.y 
    // The above three consitute a spatial state index from i and j of grid
    // last term is for including time index as well.

        // return value when full spatial grid was used
        // return (blockIdx.x + (blockIdx.y*gridDim.x) + (T*gridDim.x*gridDim.y) ); 
    
    // return value for chunks concept
    return sp_id + (T*ncells);
}


__device__ long long int state1D_from_ij(int32_t*  posid, int32_t T, int32_t gsize){
    // posid = {i , j}
    // state id = j + i*dim(i) + T*dim(i)*dim(j)

        // return value when full spatial grid was used
        // return (posid[1] + posid[0]*gridDim.x + (T*gridDim.x*gridDim.y) ) ; 

    // return value for chunks concept
    return (posid[1] + posid[0]*gsize + (T*gsize*gsize)*1LL ) ; 

}


__device__ int32_t get_rzn_id(){
    return (blockIdx.z * blockDim.x)  + threadIdx.x;
}

__device__ void get_posids_from_sp_id(long long int sp_id, int gsize, int32_t* posids){

    posids[0] = sp_id/gsize;
    posids[1] = sp_id%gsize;
    return;
}

__device__ long long int get_sp_id(){
    // sp_id: 1d spatial id ranging from 0 to ncells
    int i = blockIdx.y;
    int j = blockIdx.x;
    long long int sp_id = j + (i*gridDim.x)*1LL;
    return sp_id;
}


__device__ void get_posids_relS2_0(int32_t m, int32_t* posids_S1, int32_t* posids_relS2_0){
    // m*m is size of neighbour grid
    // returns i,j index of upper left corner of neighbour grid
    int32_t i1 = posids_S1[0];
    int32_t j1 = posids_S1[1];   
    int32_t del = (m-1)/2;
    posids_relS2_0[0] =  i1 - del;
    posids_relS2_0[1] =  j1 - del;
    return;
}


__device__ long long int get_rel_sp_id2(int32_t m, int32_t* posids_S2, int32_t* posids_relS2_0){
    // returns relative sp_id for S2 in neighbour grid

    int32_t del_i = posids_S2[0] - posids_relS2_0[0]; // i2 - rel_i0
    int32_t del_j = posids_S2[1] - posids_relS2_0[1]; // j2 - rel_j0
    long long int rel_sp_id2;
    // if S2 outstde neighbor grid, assign special value to rel_sp_id2
    // this will help keep count of no. of S2s that lie outside neighb grid
    // ideally there should be 0 such S2s
    if (del_i < m && del_j < m)
        rel_sp_id2 = del_j + (m*del_i);
    else 
        rel_sp_id2 = m*m;               

    return rel_sp_id2;
}



__device__ long long int get_sp_id2_from_rel_sp_id2(int32_t m, int32_t gsize, 
                                    long long int rel_sp_id2, int32_t* posids_relS2_0){
    // returns Sp_id2 from rel_sp_id2
    long long int sp_id2;
    int32_t del_i = rel_sp_id2/m;
    int32_t del_j = rel_sp_id2%m;

    int32_t i2 = del_i + posids_relS2_0[0];
    int32_t j2 = del_j + posids_relS2_0[1];

    sp_id2 = j2 + gsize*i2;

    return sp_id2;
}



__device__ bool is_edge_state(int32_t i, int32_t j){
    // n = gsize -1 that is the last index of the domain assuming square domain
    int32_t n = gridDim.x - 1;
    if (i == 0 || i == n || j == 0 || j == n ) 
        return true;
    else 
        return false;
}


__device__ bool is_in_obstacle(int sp_id, int T, long long int ncells, int* all_mask_mat){
    //returns true if obstacle is present in state T,i,j

    long long int mean_id = state1D_from_spid(T, sp_id, ncells);
    return(all_mask_mat[mean_id] == 1 );

}


__device__ bool is_terminal(int32_t i, int32_t j, float* params){
    // terminal state indices (of UL corner of terminal subgrid if term_subgrid_size>1)
    int32_t i_term = params[8];         
    int32_t j_term = params[9];
    int tsgsize = params[12]; //term_subgrid_size

    if( (i >= i_term && i < i_term + tsgsize)  && (j >= j_term && j < j_term + tsgsize) )
        return true;
    else return false;
}


__device__ bool my_isnan(int s){
    // By IEEE 754 rule, NaN is not equal to NaN
    return s != s;
}


__device__ void get_xypos_from_ij(int32_t i, int32_t j, int32_t gsize ,float* xs, float* ys, float* x, float* y){
    *x = xs[j];
        // *y = ys[gridDim.x - 1 - i];
    *y = ys[gsize - 1 - i];

    return;
}


__device__ long long int get_sp_id_from_posid(int32_t* posids, int32_t gsize){
    // gives sp_id from posids (i,j)
    return posids[1] + gsize*posids[0]*1LL ;
}


__device__ float get_angle_in_0_2pi(float theta){
    float f_pi = 3.141592;
    if (theta < 0)
        return theta + (2*f_pi);
    else
        return theta;
}






/*
----- move() and reward_functions() moverd to move_and_rewards.h/cu-----
*/

__device__ float calculate_one_step_reward(float ac_speed, float ac_angle, float rad1, float rad2, float* params){

    int method = params[13];
    float alpha = params[19];
    float Cr = 1;       // coeffecient for radaition term
    float Cf = 1;       // coeffecient for energy consumtion
    float Ct = 0.01;   // small coeffecient for time for to prevent lazy start
    float dt = params[4];
    float energy_reward, time_reward;

    if (method == 0)    //time
        return -dt;

    else if (method == 1){   //energy1
        return -(Cf*ac_speed*ac_speed + Ct)*dt;
    } 

    else if (method == 2){  //energy2: maximise (collection-consumption)
        return ((Cr*(rad2 + rad1)/2) - (Cf*ac_speed*ac_speed) - Ct)*dt;
    }

    else if (method == 3){  // energy3: maximise (collection)
        return ((Cr*(rad2 + rad1)/2)- Ct)*dt;
    }

    else if (method == 4){  // custom1: energy consumption vs time optimal front
        energy_reward = -(Cf*(ac_speed*ac_speed)+Ct)*dt;
        time_reward = -dt;
        return alpha*energy_reward + (1-alpha)*time_reward;
    }

    else if (method == 5){  // custom2: energy (collected-consumed) vs time
        energy_reward = (((rad2 + rad1)/2) - ac_speed*ac_speed - Ct)*dt;
        time_reward = -dt;
        return alpha*energy_reward + (1-alpha)*time_reward;
    }

    else if (method == 6){  // custom3: energy collected vs time
        energy_reward = (((rad2+rad1)/2) - Ct)*dt;
        time_reward = -dt;
        return alpha*energy_reward + (1-alpha)*time_reward;
    }


    else 
        return 0;   

}


__device__ void move(float ac_speed, float ac_angle, float vx, float vy, int32_t T, float* xs, float* ys, int32_t* posids, float* params, float* r ){
    int32_t gsize = params[0];
    int32_t n = params[0] - 1;      // gsize - 1
    // int32_t num_actions = params[1];
    // int32_t nrzns = params[2];
    // float F = params[3];
    int32_t nt = params[10];
    float F = ac_speed;
    float dt = params[4];
    float r_outbound = params[5];
    float r_terminal = params[6];
    // int32_t nT = params[10];
    float Dj = fabsf(xs[1] - xs[0]);
    float Di = fabsf(ys[1] - ys[0]);
    int32_t i0 = posids[0];
    int32_t j0 = posids[1];
    float vnetx = F*cosf(ac_angle) + vx;
    float vnety = F*sinf(ac_angle) + vy;
    float x, y;
    get_xypos_from_ij(i0, j0, gsize, xs, ys, &x, &y); // x, y stores centre coords of state i0,j0
    float xnew = x + (vnetx * dt);
    float ynew = y + (vnety * dt);
    // float r_step = 0;
    *r = 0;         // intiilaise r with 0

    if (xnew > xs[n])
        {
            xnew = xs[n];
            *r += r_outbound;
        }
    else if (xnew < xs[0])
        {
            xnew = xs[0];
            *r += r_outbound;
        }
    if (ynew > ys[n])
        {
            ynew =  ys[n];
            *r += r_outbound;
        }
    else if (ynew < ys[0])
        {
            ynew =  ys[0];
            *r += r_outbound;
        }
    int32_t xind, yind;
    float remx = remainderf((xnew - xs[0]), Dj);
    float remy = remainderf(-(ynew - ys[n]), Di);
    xind = ((xnew - xs[0]) - remx)/Dj;
    yind = (-(ynew - ys[n]) - remy)/Di;
    if ((remx >= 0.5 * Dj) && (remy >= 0.5 * Di))
        {
            xind += 1;
            yind += 1;
        }
    else if ((remx >= 0.5 * Dj && remy < 0.5 * Di))
        {
            xind += 1;
        }
    else if ((remx < 0.5 * Dj && remy >= 0.5 * Di))
        {
            yind += 1;
        }
    if (!(my_isnan(xind) || my_isnan(yind)))
        {   
            // update posids
            posids[0] = yind;
            posids[1] = xind;
            if (is_edge_state(posids[0], posids[1]))  //line 110
                {
                    *r += r_outbound;
                }
        
        }

    if (is_terminal(posids[0], posids[1], params))
        {
            *r += r_terminal;
        }
    else{
            // //reaching any state in the last timestep which is not terminal is penalised
            // if (T == nt-2)
            //     *r += r_outbound; 
            //reaching any state in the last timestep which is not terminal is penalised
            if (T > nt-20)
            *r += 0.05*r_outbound; 
        }

    }




__device__ void extract_velocity(int32_t* posids, long long int sp_id, long long int ncells, float* vx, float* vy,
    int32_t T, float* all_u_mat, float* all_v_mat, float* all_ui_mat, 
    float* all_vi_mat, float* all_Yi, float* params){


    int32_t nrzns = params[2];
    int32_t nmodes = params[7];    
    int32_t gsize = params[0];          

    long long int sp_uvi, str_uvi, sp_Yi; //startpoints and strides for accessing all_ui_mat, all_vi_mat and all_Yi
    // int str_Yi;
    float sum_x = 0;
    float sum_y = 0;
    float vx_mean, vy_mean;
    //thread index. also used to access resultant vxrzns[nrzns, gsize, gsize]
    int32_t idx = get_thread_idx();
    //rzn index to identify which of the 5k rzn it is. used to access all_Yi.
    int32_t rzn_id = get_rzn_id() ;
    //mean_id is the index used to access the flattened all_u_mat[t,i,j].
    long long int mean_id = state1D_from_spid(T, sp_id, ncells);
    //to access all_ui_mat and all_vi_mat
    //str_uvi = gridDim.x * gridDim.y;
    // sp_uvi = (T * nmodes * str_uvi) + (gridDim.x * blockIdx.y) + (blockIdx.x);
    str_uvi = gsize*gsize*1LL;
    sp_uvi = (T * nmodes * str_uvi) + (gsize * posids[0]) + (posids[1]);

    // to access all_Yi
    sp_Yi = (T * nrzns * nmodes * 1LL) + (rzn_id * nmodes);
    vx_mean = all_u_mat[mean_id];
    for(int i = 0; i < nmodes; i++)
    {
    sum_x += all_ui_mat[sp_uvi + (i*str_uvi)]*all_Yi[sp_Yi + i];
    }
    vy_mean = all_v_mat[mean_id];
    for(int i = 0; i < nmodes; i++)
    {
    sum_y += all_vi_mat[sp_uvi + (i*str_uvi)]*all_Yi[sp_Yi + i];
    }

    *vx = vx_mean + sum_x;
    *vy = vy_mean + sum_y;

    return;
}


__device__ void extract_radiation(long long int sp_id, int32_t T, long long int ncells, 
                                float* all_s_mat, float* rad){
    // for DETERMINISTIC radiation (scalar) field
    // extract radiation (scalar) from scalar matrix 
    
    long long int mean_id = state1D_from_spid(T, sp_id, ncells);
    *rad = all_s_mat[mean_id];

    return;
}


__device__ bool is_within_band(int i, int j, int i1, int j1, int i2, int j2, float* xs, float* ys, int gsize){
    //returns true if i,j are within the band connecticng cells i1,j1 and i2,j2

    if(i1==i2 || j1==j2){
        return true;
    }
    else{
        float x, y, x1, y1, x2, y2;
        float cell_diag = fabsf(xs[1]-xs[0])*1.414213;
        get_xypos_from_ij(i, j, gsize, xs, ys, &x, &y); // x, y stores centre coords of state i0,j0
        get_xypos_from_ij(i1, j1, gsize, xs, ys, &x1, &y1); 
        get_xypos_from_ij(i2, j2, gsize, xs, ys, &x2, &y2);
        float A = (y2-y1)/(x2-x1);
        float B = -1;
        float C = y1 - (A*x1);
        float dist_btw_pt_line = fabsf(A*x + B*y + C)/sqrtf((A*A) + (B*B));
        
        if (dist_btw_pt_line < cell_diag)
            return true;
        else
            return false;
    }
}


__device__ bool goes_through_obstacle(long long int sp_id1, long long int sp_id2, int T, 
                                        long long int ncells, int* D_all_mask_mat, 
                                        float* xs, float* ys, float* params){

    // returns true if the transition involves going through obstacle

    bool possible_collision = false;
    int posid1[2];
    int posid2[2];
    int gsize = params[0];
    long long int sp_id;
    get_posids_from_sp_id(sp_id1, gsize, posid1);
    get_posids_from_sp_id(sp_id2, gsize, posid2);
    int imin = min(posid1[0], posid2[0]);
    int imax = max(posid1[0], posid2[0]);
    int jmin = min(posid1[1], posid2[1]);
    int jmax = max(posid1[1], posid2[1]);
    
    for(int i=imin; i<=imax; i++){
        for(int j=jmin; j<=jmax; j++){
            if(!(i==posid1[0]&&j==posid1[1])){
                sp_id = j + gsize*i*1LL ;
                if ( is_in_obstacle(sp_id, T, ncells, D_all_mask_mat) || is_in_obstacle(sp_id, T+1, ncells, D_all_mask_mat)){
                    if (is_within_band(i, j, posid1[0], posid1[1], posid2[0], posid2[1], xs, ys, gsize) == true){
                        possible_collision = true;
                        return true;
                    }
                }
            }
        }
    }
    
    return possible_collision;
}




//test: changer from float* to float ac_angle
__global__ void transition_calc(float* T_arr, long long int ncells, 
                            float* all_u_mat, float* all_v_mat, float* all_ui_mat, float* all_vi_mat, float* all_Yi,
                            float* D_all_s_mat, int* D_all_mask_mat,
                            float ac_speed, float ac_angle, float* xs, float* ys, float* params, float* sumR_sa, 
                            float* results){
                            // resutls directions- 1: along S2;  2: along S1;    3: along columns towards count
    int32_t gsize = params[0];          // size of grid along 1 direction. ASSUMING square grid.
    int32_t nrzns = params[2]; 
    float r_outbound = params[5];        
    // int32_t is_stationary = params[11];
    int32_t T = (int32_t)T_arr[0];          // current timestep
    int32_t idx = get_thread_idx();
    long long int res_idx;
    float vx, vy, rad1, rad2;
    long long int sp_id = get_sp_id();      //sp_id is space_id. S1%(gsize*gsize)
    long long int sp_id2;
    long long int rel_sp_id2;
    int32_t posids_relS2_0[2];
    int32_t posids_S1[2];
    int32_t m = (int32_t) params[18];
    int32_t Nb = (m*m) + 1;
    float one = 1.0;
    
    if(idx < gridDim.x*gridDim.y*nrzns && sp_id < ncells) //or idx < arr_size
    {
        // int32_t posids[2] = {(int32_t)blockIdx.y, (int32_t)blockIdx.x};    //static declaration of array of size 2 to hold i and j values of S1. 
        int32_t posids[2];    //static declaration of array of size 2 to hold i and j values of S1. 
        get_posids_from_sp_id(sp_id, gsize, posids);
        get_posids_from_sp_id(sp_id, gsize, posids_S1);
        int32_t rzn_id = get_rzn_id();
        
        //  Afer move() these will be overwritten by i and j values of S2
        float r=0;              // to store immediate reward
        float r_step;

        
        extract_velocity(posids, sp_id, ncells, &vx, &vy, T, all_u_mat, all_v_mat, all_ui_mat, all_vi_mat, all_Yi, params);
        extract_radiation(sp_id, T, ncells, D_all_s_mat, &rad1);
        
        // if s1 not terminal
        if (is_terminal(posids[0], posids[1], params) == false){
            // if s1 not in obstacle
            if (is_in_obstacle(sp_id, T, ncells, D_all_mask_mat) == false){

                // moves agent and adds r_outbound and r_terminal to r
                move(ac_speed, ac_angle, vx, vy, T, xs, ys, posids, params, &r);
                sp_id2 = get_sp_id_from_posid(posids, gsize);
                extract_radiation(sp_id2, T+1, ncells, D_all_s_mat, &rad2);
                
                // adds one step-reward based on method. mehthod is available in params
                r_step = calculate_one_step_reward(ac_speed, ac_angle, rad1, rad2, params);
                r += r_step;

                // if S2 is an obstacle cell. then penalise with r_outbound
                // if (is_in_obstacle(sp_id2, T+1, ncells, D_all_mask_mat) == true )
                //     r = r_outbound;
                if (goes_through_obstacle(sp_id, sp_id2, T, ncells, D_all_mask_mat, xs, ys, params) == true)
                    r = r_outbound;
            }
            // if s1 is in obstacle, then no update to posid
            else
                r = r_outbound;
        }
  
        get_posids_relS2_0(m, posids_S1, posids_relS2_0);
        rel_sp_id2 = get_rel_sp_id2(m, posids, posids_relS2_0);
        res_idx = sp_id*Nb + rel_sp_id2;
        float b = atomicAdd(&results[res_idx], one);

        //writing to sumR_sa. this array will later be divided by nrzns, to get the avg
        float a = atomicAdd(&sumR_sa[sp_id], r); 

        __syncthreads();

    }//if ends
    return;
}



__global__ void compute_mean(float* D_master_sumRsa_arr, int size, int nrzns) {
    // computes mean
    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
    if (tid < size)
        D_master_sumRsa_arr[tid] =  D_master_sumRsa_arr[tid]/nrzns;

    return;
}



__global__ void count_kernel(float* D_master_S2_arr_ip, int nrzns, unsigned long long int* num_uq_s2_ptr) {
    // D_master_S2_arr_ip contains count of relS2s for S1s for a given action
    // This kernel counts no. of nnz elements for a given S1
    // This is needed for getting total nnz to initiliase COO matrix
    // ncells is gridDim,  i.e. we have ncells blocks in grid
    // Nb is blockDim, i.e we have Nb threads in block
    
    int ncells = gridDim.x;  // == ncells == ncells
    int Nb = blockDim.x;
    long long int tid = (blockIdx.x*Nb) + threadIdx.x;
    int idx = blockIdx.x;
    float nnz;
    unsigned long long int one = 1.0;

    if ((tid < ncells*Nb) && (threadIdx.x != Nb-1)){  // tid < Nb*ncells
        if (D_master_S2_arr_ip[tid] != 0){
            nnz = atomicAdd(&num_uq_s2_ptr[idx], one);
        }
    }
    return;
}


__global__ void reduce_kernel(float* D_master_S2_arr_ip, int t, int Nb, int m,
                            long long int ncells, int nrzns, int gsize, 
                            long long int* D_coo_s1_arr, long long int* D_coo_s2_arr, 
                            float* D_coo_cnt_arr, unsigned long long int* num_uq_s2_ptr, unsigned long long int* prSum_num_uq_s2_ptr){

    long long int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
    long long int start_idx = tid*Nb; // to access tid'th threads 0-pos in ip_arr

    long long int n_uqs = num_uq_s2_ptr[tid]; //number of unique S2s for tid'th block
    long long int op_st_id = prSum_num_uq_s2_ptr[tid];   //sum of number of uniqeu S2s uptil tid'th block. to access tid'th thread's 0-pos in op_arr

    long long int ith_nuq = 0; //ranges from 0 to n_uqs , to index number between 0 and n_uqs

    long long int rel_sp_id2;
    long long int sp_id2;
    long long int S2;
    long long int sp_id1 = tid;
    float count; //first if eval will lead to else condition and do  count++ 
    int32_t posids_relS2_0[2];
    int32_t posids_S1[2];


    if (tid < ncells){

        // int32_t s1 = (tid%ncells) + (t*ncells); // TODO:xxdone change this to nbe a function of a arguments: sp_id and t
        long long int s1 = tid + (t*ncells);
        for(long long int i = 0; i< n_uqs; i++)
            D_coo_s1_arr[op_st_id + i] = s1;
        
        get_posids_relS2_0(m, posids_S1, posids_relS2_0);
        for(long long int i = 0; i< Nb-1; i++){
            count = D_master_S2_arr_ip[start_idx + i];
            if (count != 0){
                rel_sp_id2 = i;
                get_posids_from_sp_id(sp_id1, gsize, posids_S1);
                get_posids_relS2_0(m, posids_S1, posids_relS2_0);
                sp_id2 = get_sp_id2_from_rel_sp_id2(m, gsize, 
                    rel_sp_id2, posids_relS2_0);
                S2 = state1D_from_spid(t+1, sp_id2, ncells);
                D_coo_s2_arr[op_st_id + ith_nuq] = S2;         // store old_s2 value in the [.. + ith] position
                D_coo_cnt_arr[op_st_id + ith_nuq] = count/nrzns;   // store prob value in the [.. + ith] position
                ith_nuq++;                                      // increment i
            }
        }

   }
   return;
}


template<typename dType>
void print_array(dType* array, int num_elems,std::string array_name, std::string end){
    std::cout << array_name << std::endl;
    for(int i = 0; i < num_elems; i++)
        std::cout << array[i] << " " << end;
    std::cout << std::endl;
}




std::string get_prob_name(int num_ac_speeds, int num_ac_angles, int i_term, int j_term,
                            int tsg_size){

    std::string s_n_ac_sp = std::to_string(num_ac_speeds);
    std::string s_n_ac_ac = std::to_string(num_ac_angles);
    std::string s_i = std::to_string(i_term);
    std::string s_j = std::to_string(j_term);
    std::string s_tsg = std::to_string(tsg_size);

    std::string name = "a" + s_n_ac_sp + "x" + s_n_ac_ac + "_" 
                        + "i" + s_i + "_" "j" + s_j + "_"
                        + "ref" + s_tsg;

    return name;
}







void build_sparse_transition_model_at_T_at_a(int t, int action_id, int bDimx, thrust::device_vector<float> &D_tdummy, 
                                float* D_all_u_arr, float* D_all_v_arr, float* D_all_ui_arr,
                                float*  D_all_vi_arr, float*  D_all_yi_arr,
                                float* D_all_s_arr, int* D_all_mask_arr,
                                thrust::device_vector<float> &D_params, 
                                thrust::host_vector<float> &H_params,
                                thrust::device_vector<float> &D_xs, 
                                thrust::device_vector<float> &D_ys, 
                                float** H_actions,
                                thrust::host_vector<int32_t> &H_coo_len_per_ac,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS1,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS2,
                                thrust::host_vector<float>* H_Aarr_of_cooProb,
                                thrust::host_vector<float>* H_Aarr_of_Rs
                                );

void build_sparse_transition_model_at_T_at_a(int t, int action_id, int bDimx, thrust::device_vector<float> &D_tdummy, 
                                float* D_all_u_arr, float* D_all_v_arr, float* D_all_ui_arr,
                                float*  D_all_vi_arr, float*  D_all_yi_arr,
                                float* D_all_s_arr, int* D_all_mask_arr,
                                thrust::device_vector<float> &D_params, 
                                thrust::host_vector<float> &H_params,
                                thrust::device_vector<float> &D_xs, 
                                thrust::device_vector<float> &D_ys, 
                                float** H_actions,
                                thrust::host_vector<int32_t> &H_coo_len_per_ac,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS1,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS2,
                                thrust::host_vector<float>* H_Aarr_of_cooProb,
                                thrust::host_vector<float>* H_Aarr_of_Rs
                                ){


    int gsize = (int) H_params[0];
    int num_actions =  (int)H_params[1];
    int nrzns = (int) H_params[2];
    int nt = (int) H_params[10];
    int m = (int)H_params[18];
    int Nb = (m*m) + 1; //+1 is to store no. of S2s not lying in nieghbour_array. Ideally it should have 0
                               
    // raw pointer casts
    float* D_T_arr = thrust::raw_pointer_cast(&D_tdummy[0]);
    float* xs = thrust::raw_pointer_cast(&D_xs[0]);
    float* ys = thrust::raw_pointer_cast(&D_ys[0]);
    float* params = thrust::raw_pointer_cast(&D_params[0]);

    //Define Kernel launch parameters for transition calculation kernel
    int DimGrid_z = (nrzns/bDimx)+1;
    if (nrzns % bDimx == 0)
        DimGrid_z = (nrzns/bDimx);
    
    // initialse master S2 array
    thrust::device_vector<float> D_master_S2_vector(ncells * Nb, 0);
    float* D_master_S2_arr = thrust::raw_pointer_cast(&D_master_S2_vector[0]);
    
    // initialise master sum_Rsa array - sumRsa's 
    // Important to initialise it with 0
    thrust::device_vector<float> D_master_sumRsa_vector(ncells, 0);
    float* D_master_sumRsa_arr = thrust::raw_pointer_cast(&D_master_sumRsa_vector[0]);

    // define kerel block and grid configuration
    dim3 DimGrid(gsize, gsize, DimGrid_z);
    dim3 DimBlock(bDimx, 1, 1);

    float ac_speed = H_actions[action_id][0];
    float ac_angle = H_actions[action_id][1];

    // launch kernel for @a @t
    transition_calc<<< DimGrid, DimBlock >>> (D_T_arr, 
        ncells, D_all_u_arr, D_all_v_arr, D_all_ui_arr, D_all_vi_arr, D_all_yi_arr,
        D_all_s_arr, D_all_mask_arr,
        ac_speed, ac_angle, xs, ys, params, D_master_sumRsa_arr, 
        D_master_S2_arr);

    hipDeviceSynchronize();

    // checks
    // std::cout << "D_xs= " << std::endl;
    // for (int i = 0; i< 10; i++)
    //     std::cout << D_xs[i] << " " ;

    // if (t == nt-2){
    //     std::cout << "t = " << t << "\n nt = " << nt << "\n" ; 
    //     std::cout<<"gisze= " << gsize << std::endl;
    //     std::cout<<"DimGrid_z = " << DimGrid_z << std::endl;
    //     std::cout<<"bDimx = " <<  bDimx << std::endl;
    // }

    // // CHECK copy data back to host for check
    // std::cout << "a" << n <<"\n vx at s1=0: " << D_params[31] << std::endl;
    // std::cout <<"\n vx at s1=0: " << D_params[30] << std::endl;
    // std::cout << "----a" << n <<"\n";
    // std::cout <<"pre move " << "\n";
    // std::cout<<"r1\n"<< D_params[23] << "," << D_params[24] << std::endl;
    // std::cout<<"r2\n"<< D_params[25] << "," << D_params[26] << std::endl;
    // std::cout <<"post move " << "\n";
    // std::cout<<"r1\n"<< D_params[27] << "," << D_params[28] << std::endl;
    // std::cout<<"r2\n"<< D_params[29] << "," << D_params[30] << std::endl;

    // thrust::copy(D_master_S2_vector.begin() + n*arr_size, D_master_S2_vector.begin() + (n+1)*arr_size, H_S2_vec.begin());
    // thrust::copy(D_master_sumRsa_vector.begin() + n*ncells, D_master_sumRsa_vector.begin() + (n+1)*ncells, H_sumR_sa.begin());
    // std::cout << "post kernel" << std::endl;
    // for(int i = 0; i < 10; i ++)
    //     std::cout << H_sumR_sa[i] << std::endl;
    // for(int i = 0; i < 10; i ++)
    //     std::cout << H_S2_vec[i] << std::endl;
    

    int Nthreads = D_master_sumRsa_vector.size();
    assert(Nthreads == ncells);

    int threads_per_block = 64;
    int blocks_per_grid = (Nthreads/threads_per_block) + 1;
    assert( blocks_per_grid * threads_per_block >= Nthreads);
    
    compute_mean<<< blocks_per_grid, threads_per_block >>>(D_master_sumRsa_arr, Nthreads, nrzns);

    // TODO: in optimazation phase move this line after initilisation num_uq_S2 vectors.
    // hipDeviceSynchronize();

    //initialising vectors for counting nnzs or number of uniqe S2s for S1s
    thrust::device_vector<unsigned long long int> D_num_uq_s2(ncells,0);
    thrust::device_vector<unsigned long long int> D_prSum_num_uq_s2(ncells);
    unsigned long long int* num_uq_s2_ptr = thrust::raw_pointer_cast(&D_num_uq_s2[0]);
    unsigned long long int* prSum_num_uq_s2_ptr = thrust::raw_pointer_cast(&D_prSum_num_uq_s2[0]);
    //one thread per element
    // count no. of ug unique S2 for each S1 and fill in num_uq_s2
    count_kernel<<<ncells, Nb>>>(D_master_S2_arr, nrzns, num_uq_s2_ptr);
    hipDeviceSynchronize();

        //CHECKs
        // std::cout << "D_num_uq_s2_pc\n";
        // int tempflag = 0;
        // int tempnum;
        // int cnt2 = 0;
        // int cnt1 = 0;
        // for (int i =0; i < efCszNa; i++){
        //     tempnum = D_num_uq_s2_pc[i];
        //     if (tempnum == 1)
        //         cnt1++;
        //     else if (tempnum == 2)
        //         cnt2++;
        //     else
        //         std::cout << " --------------------------- WRONG-----------\n";
        // }
        // std::cout << "cnt1 = " << cnt1 << "\ncnt2 = " << cnt2 <<"\n";


    // calc nnz: number of non zero elements(or unique S2s) for a given S1 and action
    long long int nnz = thrust::reduce(D_num_uq_s2.begin(), D_num_uq_s2.end(), (float) 0, thrust::plus<float>());
    // get prefix sum of D_num_uq_s2. This helps threads to access apt COO indices in reduce_kernel
    thrust::exclusive_scan(D_num_uq_s2.begin(), D_num_uq_s2.end(), D_prSum_num_uq_s2.begin());
    // std::cout << "nnz = " << nnz<< "\n";

    //initilise coo arrays (concated across actions)
    thrust::device_vector<long long int> D_coo_s1(nnz);
    thrust::device_vector<long long int> D_coo_s2(nnz);
    thrust::device_vector<float> D_coo_count(nnz); // TODO: makde this int32_t and introduce another array for prob
    long long int* D_coo_s1_arr = thrust::raw_pointer_cast(&D_coo_s1[0]);
    long long int* D_coo_s2_arr = thrust::raw_pointer_cast(&D_coo_s2[0]);
    float* D_coo_cnt_arr = thrust::raw_pointer_cast(&D_coo_count[0]);

    Nthreads = ncells;
    assert(Nthreads == ncells);
    threads_per_block = 64;
    blocks_per_grid = (Nthreads/threads_per_block) + 1;
    // reduce operation to fill COO arrays
    reduce_kernel<<<blocks_per_grid, threads_per_block>>>(D_master_S2_arr, t, Nb, m,
                                ncells, nrzns, gsize, D_coo_s1_arr, D_coo_s2_arr, D_coo_cnt_arr, 
                                num_uq_s2_ptr, prSum_num_uq_s2_ptr);
    hipDeviceSynchronize();


    // nnz should be filled in a global array 
    H_coo_len_per_ac[action_id] = nnz;
    // Copy Device COO rusults to Host COO vectors across actions and append vectors across time
    assert(action_id >=0);
    H_Aarr_of_cooS1[action_id].insert(H_Aarr_of_cooS1[action_id].end(), D_coo_s1.begin(), D_coo_s1.end());
    H_Aarr_of_cooS2[action_id].insert(H_Aarr_of_cooS2[action_id].end(), D_coo_s2.begin(), D_coo_s2.end());
    H_Aarr_of_cooProb[action_id].insert(H_Aarr_of_cooProb[action_id].end(), D_coo_count.begin(), D_coo_count.end());
    H_Aarr_of_Rs[action_id].insert(H_Aarr_of_Rs[action_id].end(), D_master_sumRsa_vector.begin(), D_master_sumRsa_vector.end());

        //checks

        // print_device_vector(D_coo_s1, 0, 10, "D_coo_s1", " ", 0);
        // print_device_vector(D_coo_s2, 0, 10, "D_coo_s2", " ", 0);

        // std::cout << "H_coo_len_per_ac" << std::endl;
        // for (int n = 0; n < num_actions; n++)
        //   std::cout << H_coo_len_per_ac[n] << std::endl;

        // std::cout << "H_Aarr_of_cooS1" << std::endl;
        // for (int n = 0; n < num_actions; n++){
        //     for (int i = 0; i < H_Aarr_of_cooS1[n].size(); i++)
        //         std::cout << H_Aarr_of_cooS1[n][i] << " , " << H_Aarr_of_cooS2[n][i] << " , " << H_Aarr_of_cooProb[n][i] << std::endl;
        //     std::cout << std::endl;
        // }

        // std::cout << "H_Aarr_of_Rs" << std::endl;
        // for (int n = 0; n < num_actions; n++){
        //     for (int i = 0; i < ncells; i++)
        //         std::cout << H_Aarr_of_Rs[n][i] << std::endl;
        //     std::cout << std::endl;
        // }

        // // array of num_actions decive_vvectors for sum_Rsa_vec
        // // initialasation with 0 is important. because values are added to this
        // thrust::host_vector<float> H_arr_sumR_sa[num_actions];
        // for(int n = 0; n < num_actions; n++){
        //     H_arr_sumR_sa[n] = thrust::host_vector<float>(nnz[i]);
    

}





int get_reward_type(std::string prob_type){
    // returns 
    // 0 for time
    // 1 for energy1
    // 2 for energy2
    // 3 for energy3
    // 4 for custom1

    if (prob_type == "time")
        return 0;
    else if (prob_type == "energy1")
        return 1;
    else if (prob_type == "energy2")
        return 2;
    else if (prob_type == "energy3")
        return 3;
    else if (prob_type == "custom1")
        return 4;
    else if (prob_type == "custom2")
        return 5;
    else if (prob_type == "custom3")
        return 6;
    else
        return -1;
}


// ------------------------------- MAIN ()  ---------------------------------




int main(int argc, char *argv[]){

    // add input arguement part
    // -------------------- input data starts here ---------------------------------


    #include "input_to_build_model.h"

    auto start_build = high_resolution_clock::now();
    auto end_build = high_resolution_clock::now();
    auto build_t = duration_cast<microseconds>(end_build - start_build);

    float alpha = alpha_header;

    if(argc>1){
        std::cout << alpha << " and " << argv[0] << " and " << argv[1] << "\n";
        alpha = std::stof(argv[1]);
        alpha = alpha/1000;
        // std::stringstream convert{ argv[2] };
        // if(!(convert >> alpha)) alpha = 0;
        std::cout << alpha << "\n";
    }

    std::cout << argc << "\n";
 
    std::cout << alpha << " and " << argv[0] << " and " << argv[1] << "\n";
    int reward_type = get_reward_type(prob_type);
    std::cout << "Reward type: " << reward_type << "\n";

    // define full problem name and print them to a temporary file
    // the temp file will be read by python scripts for conversion
    std::string prob_specs = get_prob_name(num_ac_speeds, num_ac_angles, i_term, 
                                            j_term, term_subgrid_size);
    std::string op_Fname_upto_prob_name = "data_modelOutput/" + prob_type + "/"
                                 + prob_name + "/" ;
    std::string op_FnamePfx = op_Fname_upto_prob_name + prob_specs + "/"; //path for storing op npy data.

    std::string op_Fname_withAlpha = op_FnamePfx + std::to_string(alpha) +"/";

   make_dir(op_Fname_upto_prob_name);
    make_dir(op_FnamePfx);

    if(reward_type>3){
        op_FnamePfx = op_Fname_withAlpha;
        make_dir(op_FnamePfx);
    }

    std::ofstream fout("temp_modelOp_dirName.txt");
    fout << prob_type << "\n";
    fout << prob_name << "\n";
    fout << prob_specs << "\n";
    fout << std::to_string(alpha) << "\n";
    fout << op_FnamePfx;
    fout.close();

    // TODO:  Make sure files are stored in np.float32 format
    std::string data_path = "data_input/" + prob_name + "/";
    std::string all_u_fname = data_path + "all_u_mat.npy";
    std::string all_v_fname = data_path + "all_v_mat.npy";
    std::string all_ui_fname = data_path + "all_ui_mat.npy";
    std::string all_vi_fname = data_path + "all_vi_mat.npy";
    std::string all_yi_fname = data_path + "all_Yi.npy";
    std::string all_s_fname = data_path + "all_s_mat.npy";
    std::string all_mask_fname = data_path + "obstacle_mask.npy"; //this file stored in int32


    // -------------------- input data ends here ---------------------------------

    

    // make directory for storing output data from this file
    make_dir(op_Fname_upto_prob_name);
    make_dir(op_FnamePfx);

    auto start_build_only = high_resolution_clock::now();

    int all_u_n_elms;
    int all_v_n_elms;
    int all_ui_n_elms;
    int all_vi_n_elms;
    int all_yi_n_elms;
    int all_s_n_elms;
    int all_mask_n_elms;

    cnpy::NpyArray all_u_cnpy = read_velocity_field_data(all_u_fname, &all_u_n_elms);
    cnpy::NpyArray all_v_cnpy = read_velocity_field_data(all_v_fname, &all_v_n_elms);
    cnpy::NpyArray all_ui_cnpy = read_velocity_field_data(all_ui_fname, &all_ui_n_elms);
    cnpy::NpyArray all_vi_cnpy = read_velocity_field_data(all_vi_fname, &all_vi_n_elms);
    cnpy::NpyArray all_yi_cnpy = read_velocity_field_data(all_yi_fname, &all_yi_n_elms);
    cnpy::NpyArray all_s_cnpy = read_velocity_field_data(all_s_fname, &all_s_n_elms);
    cnpy::NpyArray all_mask_cnpy = read_velocity_field_data(all_mask_fname, &all_mask_n_elms);


    float* all_u_mat = all_u_cnpy.data<float>();
    float* all_v_mat = all_v_cnpy.data<float>();
    float* all_ui_mat = all_ui_cnpy.data<float>();
    float* all_vi_mat = all_vi_cnpy.data<float>();
    float* all_yi_mat = all_yi_cnpy.data<float>();
    float* all_s_mat = all_s_cnpy.data<float>();
    int* all_mask_mat = all_mask_cnpy.data<int>();

    // CHECKS:
    // print_array<float>(all_u_mat, all_u_n_elms, "all_u_mat", " ");
    // print_array<float>(all_ui_mat, all_ui_n_elms,"all_ui_mat", " ");
    // print_array<float>(all_yi_mat, all_yi_n_elms,"all_yi_mat", " ");

    std::cout << "Finished reading Velocity Field Data !" << std::endl;
    assert(neighb_gsize <= gsize);
    assert((int)neighb_gsize%2 == 1);    // neighb_gsize must be odd

    //TODO: fill params in a function
    // Contains implicit casting from int32_t to float
    thrust::host_vector<float> H_params(32);
    H_params[0] = gsize;
    H_params[1] = num_actions; 
    H_params[2] = nrzns;
    H_params[3] = F;
    H_params[4] = dt;
    H_params[5] = r_outbound;
    H_params[6] = r_terminal;
    H_params[7] = nmodes;
    H_params[8] = i_term;
    H_params[9] = j_term;
    H_params[10] = nt;
    H_params[11] = is_stationary;
    H_params[12] = term_subgrid_size;
    H_params[13] = reward_type;
    H_params[14] = num_ac_speeds;
    H_params[15] = num_ac_angles;
    H_params[16] = dx;
    H_params[17] = dy;
    H_params[18] = neighb_gsize; // referred to as m in functions
    H_params[19] = alpha;

    for( int i =20; i<32; i++)
        H_params[i] = z;

    // Define grid ticks in host
    thrust::host_vector<float> H_xs(gsize, -1);
    thrust::host_vector<float> H_ys(gsize, -1);
    float* xs = thrust::raw_pointer_cast(&H_xs[0]);
    float* ys = thrust::raw_pointer_cast(&H_ys[0]);

    define_xs_or_ys(xs, dx, x0, gsize);
    define_xs_or_ys(ys, dy, y0, gsize);

    // define angles in host
    float** H_actions = new float*[num_actions];
    for(int i=0; i<num_actions; i++)
        H_actions[i] = new float[2];
    populate_actions(H_actions, num_ac_speeds, num_ac_angles, F);
    // std::cout << "CHECK:   ACTIONS:    \n";
    // for(int i=0; i<num_actions; i++){
    //     std::cout << H_actions[i][0] << ", " << H_actions[i][1] << "\n";
    // }



    //----- start copying data to device --------

    // Copy vel field data to device memory using thrust device_vector
    thrust::device_vector<float> D_all_u_vec (all_u_mat, all_u_mat + all_u_n_elms);
    thrust::device_vector<float> D_all_v_vec (all_v_mat, all_v_mat + all_v_n_elms);
    thrust::device_vector<float> D_all_ui_vec (all_ui_mat, all_ui_mat + all_ui_n_elms);
    thrust::device_vector<float> D_all_vi_vec (all_vi_mat, all_vi_mat + all_vi_n_elms);
    thrust::device_vector<float> D_all_yi_vec (all_yi_mat, all_yi_mat + all_yi_n_elms);
    thrust::device_vector<float> D_all_s_vec (all_s_mat, all_s_mat + all_s_n_elms);
    thrust::device_vector<int> D_all_mask_vec (all_mask_mat, all_mask_mat + all_mask_n_elms);


    float* D_all_u_arr = thrust::raw_pointer_cast(&D_all_u_vec[0]);
    float* D_all_v_arr = thrust::raw_pointer_cast(&D_all_v_vec[0]);
    float* D_all_ui_arr = thrust::raw_pointer_cast(&D_all_ui_vec[0]);
    float* D_all_vi_arr = thrust::raw_pointer_cast(&D_all_vi_vec[0]);
    float* D_all_yi_arr = thrust::raw_pointer_cast(&D_all_yi_vec[0]);
    float* D_all_s_arr = thrust::raw_pointer_cast(&D_all_s_vec[0]);
    int* D_all_mask_arr = thrust::raw_pointer_cast(&D_all_mask_vec[0]);


    std::cout << "Copied to Device : Velocity Field Data !" << std::endl;

    thrust::device_vector<float> D_tdummy(2,0);
    // initialise empty device vectors. These contain time-invariant data
    thrust::device_vector<float> D_params(32);
    thrust::device_vector<float> D_xs(gsize);
    thrust::device_vector<float> D_ys(gsize);

    // initialise reuseable host vectors
    thrust::host_vector<int32_t> H_coo_len_per_ac(num_actions);
    thrust::host_vector<long long int> H_Aarr_of_cooS1[(int)num_actions];
    thrust::host_vector<long long int> H_Aarr_of_cooS2[(int)num_actions];
    thrust::host_vector<float> H_Aarr_of_cooProb[(int)num_actions];
    thrust::host_vector<float> H_Aarr_of_Rs[(int)num_actions];

    //initialised with 0 size. later data from device is inserted/appended to the end of vector
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_cooS1[i] = thrust::host_vector<long long int> (0);
    }
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_cooS2[i] = thrust::host_vector<long long int> (0);
    }
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_cooProb[i] = thrust::host_vector<float> (0);
    }
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_Rs[i] = thrust::host_vector<float> (0);
    }

    // assign value to global variable
    ncells = gsize*gsize;           

    // copy data from host to device
    D_params = H_params;
    D_xs = H_xs;
    D_ys = H_ys;

    // run time loop and compute transition data for each time step
    auto start = high_resolution_clock::now(); 
    auto end = high_resolution_clock::now(); 
    auto duration_t = duration_cast<microseconds>(end - start);

    //IMP: Run time loop till nt-1. There ar no S2s to S1s in the last timestep
    for(int t = 0; t < nt-1; t++){
        std::cout << "*** Computing data for timestep, T = " << t << std::endl;
        D_tdummy[0] = t;
        start = high_resolution_clock::now(); 
            for(int action_id = 0; action_id < num_actions; action_id++){
                // std::cout << "  * action_id= " << action_id;
                
                // this function also concats coos across time.
                build_sparse_transition_model_at_T_at_a(t, action_id, bDimx, D_tdummy, D_all_u_arr, D_all_v_arr, 
                        D_all_ui_arr, D_all_vi_arr, D_all_yi_arr,
                        D_all_s_arr, D_all_mask_arr,
                        D_params, H_params, D_xs, D_ys, H_actions, 
                        H_coo_len_per_ac,
                        H_Aarr_of_cooS1, H_Aarr_of_cooS2, H_Aarr_of_cooProb,
                        H_Aarr_of_Rs);
                        //  output_data )  
            }
        end = high_resolution_clock::now(); 
        std::cout << std::endl ;
        duration_t = duration_cast<microseconds>(end - start);
        std::cout << "duration@t = "<< duration_t.count()/1e6 << "sec" << std::endl;
        std::cout << 3*H_Aarr_of_cooS1[0].size()*4*1e-6 << " MB" << std::endl;
        std::cout << std::endl << std::endl;
    }


    // fill R vectors of each action for the last time step with high negative values. 
    // this has to be done seaprately because the above loop runs till nt-1.
    /*
        TODO: 1. Verify rewards as last time step
    */
    thrust::host_vector<float> H_rewards_at_end_t(ncells, 0);
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_Rs[i].insert(H_Aarr_of_Rs[i].end(), H_rewards_at_end_t.begin(), H_rewards_at_end_t.end());
    }

    //Check
    // for (int i =0; i < num_actions; i++)
    //     std::cout << H_Aarr_of_Rs[i].size() << " ";
    

    // find nnz per action
    thrust::host_vector<long long int> H_master_PrSum_nnz_per_ac(num_actions);
    long long int DP_relv_params[2] = {ncells*nt, num_actions*1LL};

    long long int master_nnz = 0;       //running sum of nnz going across actions

    // calculate inclusive prefix sum of nnz's across actions 
    // will be used to access indeces while concatenating results across across actions
    for(int i = 0; i < num_actions; i++){
        master_nnz += H_Aarr_of_cooS1[i].size();
        H_master_PrSum_nnz_per_ac[i] = master_nnz;
    }

    unsigned long int num_DP_params = sizeof(DP_relv_params) / sizeof(DP_relv_params[0]);
    // print_array<long long int>(DP_relv_params, 2, "DP_relv_params", " ");
    // std::cout << "chek num = " << sizeof(DP_relv_params) << std::endl;
    // std::cout << "chek denom = " << sizeof(DP_relv_params[0]) << std::endl;

    //checks
    // std::cout << "total/master_nnz = " << master_nnz << std::endl;
    // std::cout << "H_Aarr_of_cooS1[i].size()" << std::endl;
    // for(int i = 0; i < num_actions; i++)
    //     std::cout << H_Aarr_of_cooS1[i].size() << std::endl;
    // print_array<long long int>(&H_Aarr_of_cooS2[0][0], 10,  "H_Aarr_of_cooS2[0]", " ");



    auto end_build_only = high_resolution_clock::now(); // end build only
    auto time_build_only = duration_cast<microseconds>(end_build_only - start_build_only);  // time build only


    // save final coo data
    thrust::host_vector<long long int> H_master_cooS1(master_nnz);
    thrust::host_vector<long long int> H_master_cooS2(master_nnz);
    thrust::host_vector<float> H_master_cooVal(master_nnz);
    thrust::host_vector<float> H_master_R(ncells*nt*num_actions, -99999); //TODO: veriffy -99999
    std::string op_FnamePfx_2 = "data_solverOutput/" + prob_type + "/"
    + prob_name + "/" + prob_specs + "/";
    std::string op_Fname_withAlpha_2 = "data_solverOutput/" + prob_type + "/"
    + prob_name + "/" + prob_specs + std::to_string(alpha) +"/";

    if(reward_type>3){
        op_FnamePfx_2 = op_Fname_withAlpha_2;
    }

    save_master_Coos_to_file(op_FnamePfx,op_FnamePfx_2, num_actions,
                                H_master_cooS1, 
                                H_master_cooS2, 
                                H_master_cooVal,
                                H_master_R,
                                H_Aarr_of_cooS1,
                                H_Aarr_of_cooS2,
                                H_Aarr_of_cooProb,
                                H_Aarr_of_Rs,
                                H_params,
                                DP_relv_params,
                                num_DP_params);


    end_build = high_resolution_clock::now();
    build_t = duration_cast<microseconds>(end_build - start_build); 


    

    std::ofstream fout_time("temp_runTime.txt");
    fout_time << build_t.count()/1e6 << "\n";

    fout_time << time_build_only.count()/1e6 << "\n";

    fout_time.close();


    return 0;
}

//------------ main ends here ------------------------------------------





void save_master_Coos_to_file(std::string op_FnamePfx, std::string op_FnamePfx_2, int num_actions,
    thrust::host_vector<long long int> &H_master_cooS1, 
    thrust::host_vector<long long int> &H_master_cooS2, 
    thrust::host_vector<float> &H_master_cooVal,
    thrust::host_vector<float> &H_master_R,
    thrust::host_vector<long long int>* H_Aarr_of_cooS1,
    thrust::host_vector<long long int>* H_Aarr_of_cooS2,
    thrust::host_vector<float>* H_Aarr_of_cooProb,
    thrust::host_vector<float>* H_Aarr_of_Rs,
    thrust::host_vector<float> &prob_params,
    long long int* DP_relv_params,
    unsigned long int num_DP_params){
    //  Convertes floats to int32 for COO row and col idxs
    //  copies from each action vector to a master vector
    //  master_coo vectors is concatation first across time, then across action
    //  ALSO, MODIFIES S1(t,i,j) to S1(t,i,j,a)

    unsigned long long int master_nnz = H_master_cooS1.size();
    unsigned long long int prob_params_size = prob_params.size();
    long long int m_idx = 0;
    int n_states = DP_relv_params[0];


    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooS1[i].size(); j++){
            // TODO: modify to include actions
            H_master_cooS1[m_idx] = H_Aarr_of_cooS1[i][j] + i*n_states;
            m_idx++;
        }
    }

    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooS2[i].size(); j++){
            H_master_cooS2[m_idx] = H_Aarr_of_cooS2[i][j];
            m_idx++;
        }
    }

    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooProb[i].size(); j++){
            H_master_cooVal[m_idx] = H_Aarr_of_cooProb[i][j];
            m_idx++;
        }
    }

    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_Rs[i].size(); j++){
            H_master_R[m_idx] = H_Aarr_of_Rs[i][j];
            m_idx++;
        }
    }

    
    std::cout << "check num_DP_params = " << num_DP_params << std::endl;
    std::cout << "op_FnamePfx= " <<  op_FnamePfx << "\n";
    
    cnpy::npy_save(op_FnamePfx + "master_cooS1.npy", &H_master_cooS1[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_cooS2.npy", &H_master_cooS2[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_cooVal.npy", &H_master_cooVal[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_R.npy", &H_master_R[0], {H_master_R.size(),1},"w");
    cnpy::npy_save(op_FnamePfx + "DP_relv_params.npy", &DP_relv_params[0], {num_DP_params,1},"w");
    cnpy::npy_save(op_FnamePfx + "prob_params.npy", &prob_params[0], {prob_params_size,1},"w");
    // cnpy::npy_save(op_FnamePfx_2 + "prob_params.npy", &prob_params[0], {prob_params_size,1},"w");
    std::cout << "saved files \n" ;

}



cnpy::NpyArray read_velocity_field_data( std::string file_path_name, int* n_elements){
    // reads numpy file from input and 
    // returns cnpy::NpyArray stucture  and also fills in num_elements in the passed reference n_elements
    // extraction in main: float* vel_data = arr.data<float>();
    // TODO: make it general. currently hard-coded for float arrays.

    //print filename
    std::cout << "file path and name:   " << file_path_name << std::endl;
    cnpy::NpyArray arr = cnpy::npy_load(file_path_name);

    //prints for checks 
    int dim = arr.shape.size();
    int num_elements = 1;
    std::cout << "shape: " ;
    for (int i = 0; i < dim; i++){
        std::cout << arr.shape[i] << " , " ;
        num_elements = num_elements*arr.shape[i];
    }
    *n_elements = num_elements;
    std::cout << std::endl << "num_elements: " << num_elements << std::endl;

    float* vel_data = arr.data<float>();
    // print check first 10 elements
    std::cout << "First 10 elements of loaded array are: " << std::endl;
    for (int i = 0; i < 10; i++)
         std::cout << vel_data[i] << "  " ;
    
    std::cout << std::endl << std::endl;

    return arr;

}

/*
--- print_device_vector()  ---
IMP: datatype has to be explicityle changed in that file
*/

// template<typename dType>
void print_device_vector( thrust::device_vector<long long int> &array, int start_id, int end_id, std::string array_name, std::string end, int method){
    std::cout << array_name << "  from id " << start_id << "  to  " << end_id << std::endl;
    if (method == 1){
        float temp = -10000000;
        for(int i = start_id; i < end_id; i++){
            if (array[i] != temp){
                std::cout << i << "\n";
                std::cout << array[i] << " " << end;
                std::cout << "\n";
                temp = array[i];
            }
        }
    }

    else if (method == 0){
        for(int i = start_id; i < end_id; i++)
            std::cout << array[i] << " " << end;
    }

    else
        std::cout << "Invalid input for argument: method";


    std::cout << std::endl;
}


void make_dir(std::string dir_name){
    int mkdir_status;
    std::string comm_mkdir = "mkdir ";
    std::string str = comm_mkdir + dir_name;
    const char * full_command = str.c_str();
    mkdir_status = system(full_command);
    std::cout << "mkdir_status = " << mkdir_status << std::endl;
}



void define_xs_or_ys(float* xs, float dx, float x0, int gsize){

    for(int i = 0; i < gsize;  i++)
        xs[i] = x0 + i*dx;
}



void populate_ac_angles(float* ac_angles, int num_ac_angles){
    //fills array with equally spaced angles in radians
    for (int i = 0; i < num_ac_angles; i++)
        ac_angles[i] = i*(2*M_PI)/num_ac_angles;
    return;
}



void populate_ac_speeds(float* ac_speeds, int num_ac_speeds, float Fmax){
    //fills array with ac_speeds
    // std::cout << "infunc CHeck- num_ac_speeds = " << num_ac_speeds << "\n";
    float delF = 0;
    if (num_ac_speeds == 1)
        ac_speeds[0] = Fmax;
    else if (num_ac_speeds > 1){
        // -----include 0 speed
        // delF = Fmax/(num_ac_speeds-1);
        // for(int i = 0; i<num_ac_speeds; i++)
        //     ac_speeds[i] = i*delF;
        // ------exclude 0 speed
        delF = Fmax/(num_ac_speeds);
        for(int i = 0; i<num_ac_speeds; i++){
            ac_speeds[i] = (i+1)*delF;
            // std::cout << ac_speeds[i] << "\n";
        }
    }
    else
        std::cout << "Invalid num_ac_speeds\n";
    
    return;
}



void populate_actions(float **H_actions, int num_ac_speeds, int num_ac_angles, float Fmax){
    // populates 2d vector with possible actions
    float* ac_angles = new float[num_ac_angles];
    populate_ac_angles(ac_angles, num_ac_angles);

    float* ac_speeds = new float[num_ac_speeds];
    populate_ac_speeds(ac_speeds, num_ac_speeds, Fmax);

    int idx;
    for (int i=0; i<num_ac_speeds; i++){
        for(int j=0; j<num_ac_angles; j++){
            idx = j + num_ac_angles*i;
            // std::cout << ac_speeds[i] << "\n";
            H_actions[idx][0] = ac_speeds[i];
            H_actions[idx][1] = ac_angles[j];
        }
    }

    return;
}